#include "hip/hip_runtime.h"
/* * * * *
 *  AzCuda_PmatApp.cu
 *  Copyright (C) 2014,2015,2017 Rie Johnson
 *
 *  This program is free software: you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation, either version 3 of the License, or
 *  (at your option) any later version.
 *
 *  This program is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License
 *  along with this program.  If not, see <http://www.gnu.org/licenses/>.
 * * * * */

#include "AzCuda_PmatApp.cuh"

#ifdef __AZ_GPU__
  extern int max_threads, max_blocks; 
  #include "AzCuda.cuh"
  #include "AzCuda_Pmat.cuh"  /* azc_config */
  static void chk_err(const char *eyec, int bb, int tt) {
    AzCuda::check_error(eyec, bb, tt);   
  }   
#else
  static bool azc_config(int num, int &bb, int &tt, const char *msg="") { return true; }
  static void chk_err(const char *eyec, int bb, int tt) {}
#endif   
  
  /*****  PmatApp  *****/
   /*---  L1L2: sqrt(x^2+d^2)-d  ---*/
  __global__ void azc_l1l2(const AzFloat *src, AzFloat *dst, int num, AzFloat del) {  
    double delsq = del*del; 
    int ix; 
    for (ix = azc_thno; ix < num; ix += azc_thnum) {
      dst[ix] = sqrt(src[ix]*src[ix] + delsq) - del; 
    }
  }

  /*---  L1L2-deriv: x/sqrt(x^2+d^2)  ---*/
  __global__ void azc_add_l1l2deriv(const AzFloat *src, AzFloat *dst, int num, AzFloat del, AzFloat coeff) {  
    double delsq = del*del; 
    int ix; 
    for (ix = azc_thno; ix < num; ix += azc_thnum) {
      dst[ix] += (src[ix]*coeff) / sqrt(src[ix]*src[ix] + delsq); 
    }
  } 
  void azccall_l1l2(const AzFloat *src, AzFloat *dst, int num, AzFloat del) {
    int bb, tt; 
    azc_config(num, bb, tt, "azccall_l1l2"); 
    azc_kernel(azc_l1l2,bb,tt)(src, dst, num, del); 
    chk_err("azccall_l1l2", bb, tt);     
  }  
  void azccall_add_l1l2deriv(const AzFloat *src, AzFloat *dst, int num, AzFloat del, AzFloat coeff) {
    int bb, tt; 
    azc_config(num, bb, tt, "azccall_add_l1l2deriv"); 
    azc_kernel(azc_add_l1l2deriv,bb,tt)(src, dst, num, del, coeff); 
    chk_err("azccall_add_l1l2deriv", bb, tt); 
  } 
  
  /*--------------------------------------------------*/  
  /*---        min(th,max(0,x)) activation         ---*/
  /*--------------------------------------------------*/  
  __global__ void azc_activate_th(int num, AzFloat *elm, AzFloat th) {
    for (int ex = azc_thno; ex < num; ex += azc_thnum) {
      if      (elm[ex] <= 0)           elm[ex] = 0; 
      else if (th > 0 && elm[ex] > th) elm[ex] = th;     
    }
  }                    
  __global__ void azc_activate_th_deriv(int num, AzFloat *elm, AzFloat th,
                             AzFloat *deriv_elm) /* must not be NULL */ {
    for (int ex = azc_thno; ex < num; ex += azc_thnum) {                             
      if (elm[ex] <= 0) {
        elm[ex] = 0; 
        deriv_elm[ex] = 0; 
      } else if (th > 0 && elm[ex] > th) {
        elm[ex] = th; 
        deriv_elm[ex] = 0; 
      } else {
        deriv_elm[ex] = 1; 
      }
    }
  }
  __global__ void azc_activate_rect(int num, AzFloat *elm) {
    for (int ex = azc_thno; ex < num; ex += azc_thnum) if (elm[ex] <= 0) elm[ex] = 0;  
  }                    
  __global__ void azc_activate_rect_deriv(int num, AzFloat *elm,
                             AzFloat *deriv_elm) /* must not be NULL */ {
    for (int ex = azc_thno; ex < num; ex += azc_thnum) {                             
      if (elm[ex] <= 0) {
        elm[ex] = 0; 
        deriv_elm[ex] = 0; 
      } else {
        deriv_elm[ex] = 1; 
      }
    }
  }
  /*--------------------------------------------------*/   
  void azccall_activate_th(AzFloat *elm, int num, AzFloat th, 
                           AzFloat *deriv_elm) /* may be NULL */ {
    /* note: single vs multi didn't matter */
    int bb, tt; 
    azc_config(num, bb, tt, "azccall_activate_th"); 
    if (th < 0) {
      if (deriv_elm == NULL) azc_kernel(azc_activate_rect,bb,tt)(num, elm); 
      else                   azc_kernel(azc_activate_rect_deriv,bb,tt)(num, elm, deriv_elm);     
    }
    else {
      if (deriv_elm == NULL) azc_kernel(azc_activate_th,bb,tt)(num, elm, th); 
      else                   azc_kernel(azc_activate_th_deriv,bb,tt)(num, elm, th, deriv_elm); 
    }
    chk_err("azccall_activate_th", bb, tt); 
  }
  
  /*--------------------------------------------------*/  
  /*---        max(0,x) if x>0; ax otherwise       ---*/
  /*--------------------------------------------------*/  
  __global__ void azc_activate_leaky_rect(int num, AzFloat *elm, AzFloat aa) {
    for (int ex = azc_thno; ex < num; ex += azc_thnum) if (elm[ex] <= 0) elm[ex] *= aa;     
  }                    
  __global__ void azc_activate_leaky_rect_deriv(int num, AzFloat *elm, AzFloat aa, 
                             AzFloat *deriv_elm) /* must not be NULL */ {
    for (int ex = azc_thno; ex < num; ex += azc_thnum) {                             
      if (elm[ex] <= 0) { elm[ex] *= aa; deriv_elm[ex] = aa; } 
      else                deriv_elm[ex] = 1; 
    }
  }
  /*--------------------------------------------------*/   
  void azccall_activate_leaky_rect(AzFloat *elm, int num, AzFloat aa, 
                           AzFloat *deriv_elm) /* may be NULL */ {
    int bb, tt; azc_config(num, bb, tt, "azccall_activate_leaky_rect"); 
    if (deriv_elm == NULL) azc_kernel(azc_activate_leaky_rect,bb,tt)(num, elm, aa); 
    else                   azc_kernel(azc_activate_leaky_rect_deriv,bb,tt)(num, elm, aa, deriv_elm); 
    chk_err("azccall_activate_leaky_rect", bb, tt); 
  }  

  /*--------------------------------------------------*/
  /*---             sigmoid activation             ---*/
  /*--------------------------------------------------*/  
  __global__ void azc_activate_log(AzFloat *elm, int num) {
    int ex; 
    for (ex = azc_thno; ex < num; ex += azc_thnum) {  
      elm[ex] = (AzFloat)1/((AzFloat)1+myexp(-elm[ex])); 
    }
  } 
  /*--------------------------------------------------*/  
  __global__ void azc_activate_log_deriv(AzFloat *elm, int num, 
                                    AzFloat *deriv_elm) {
    int ex; 
    for (ex = azc_thno; ex < num; ex += azc_thnum) {  
      AzFloat ss = (AzFloat)1/((AzFloat)1+myexp(-elm[ex])); 
      elm[ex] = ss; 
      deriv_elm[ex] = ss*(1-ss); 
    }
  } 
  /*--------------------------------------------------*/  
  void azccall_activate_log(AzFloat *elm, int num,
                           AzFloat *deriv_elm) /* may be NULL */ {
    int bb, tt; 
    azc_config(num, bb, tt, "azccall_activate_log"); 
    if (deriv_elm == NULL) azc_kernel(azc_activate_log,bb,tt)(elm, num); 
    else                   azc_kernel(azc_activate_log_deriv,bb,tt)(elm, num, deriv_elm); 
    chk_err("azccall_activate_log", bb, tt); 
  } 
  
  /*------------------------------------------------*/
  /*---             tanh activation              ---*/
  /*------------------------------------------------*/
  __global__ void azc_activate_tanh(AzFloat *elm, int num) {
    int ex; 
    for (ex = azc_thno; ex < num; ex += azc_thnum) {
      double e2 = myexp(2*elm[ex]); 
      elm[ex] = (AzFloat)((e2-1)/(e2+1));   
    }
  } 
  /*--------------------------------------------------*/  
  __global__ void azc_activate_tanh_deriv(AzFloat *elm, int num, 
                                  AzFloat *deriv_elm) {
    int ex; 
    for (ex = azc_thno; ex < num; ex += azc_thnum) {
      double e2 = myexp(2*elm[ex]); 
      elm[ex] = (AzFloat)((e2-1)/(e2+1));
      deriv_elm[ex] = 4*e2/(e2+1)/(e2+1); 
    }
  }
  /*--------------------------------------------------*/  
  void azccall_activate_tanh(AzFloat *elm, int num,
                             AzFloat *deriv_elm) /* may be NULL */ {
    int bb, tt; 
    azc_config(num, bb, tt, "azccall_activate_tanh"); 
    if (deriv_elm == NULL) azc_kernel(azc_activate_tanh,bb,tt)(elm, num); 
    else                   azc_kernel(azc_activate_tanh_deriv,bb,tt)(elm, num, deriv_elm); 
    chk_err("azccall_activate_tanh", bb, tt); 
  } 
  
  /*--------------------------------------------------*/
  /*---            softplus activation             ---*/
  /*--------------------------------------------------*/  
  __global__ void azc_activate_softplus(AzFloat *elm, int num) {
    int ex; 
    for (ex = azc_thno; ex < num; ex += azc_thnum) {
      double e2 = myexp(elm[ex]); 
      elm[ex] = (AzFloat)log(1+e2);   
    }
  } 
  /*--------------------------------------------------*/  
  __global__ void azc_activate_softplus_deriv(AzFloat *elm, int num, 
                                    AzFloat *deriv_elm) {
    int ex; 
    for (ex = azc_thno; ex < num; ex += azc_thnum) {
      double e2 = myexp(elm[ex]); 
      elm[ex] = (AzFloat)log(1+e2);  
      deriv_elm[ex] = (AzFloat)(e2/(e2+1)); 
    }
  }
  /*--------------------------------------------------*/  
  void azccall_activate_softplus(AzFloat *elm, int num, 
                                 AzFloat *deriv_elm) /* may be NULL */ {
    int bb, tt; 
    azc_config(num, bb, tt, "azccall_activate_softplus"); 
    if (deriv_elm == NULL) azc_kernel(azc_activate_softplus,bb,tt)(elm, num); 
    else                   azc_kernel(azc_activate_softplus_deriv,bb,tt)(elm, num, deriv_elm); 
    chk_err("azccall_activate_softplus", bb, tt); 
  }   

  /*--------------------------------------------------*/    
  /*------------------------------------------------*/
  __global__ void azc_truncate(AzFloat *elm, int num, AzFloat border) {
    int i; 
    for (i = azc_thno; i < num; i +=azc_thnum) {
      elm[i] = MAX(-border, MIN(border, elm[i])); 
    }
  }
  __global__ void azc_truncate_deriv(AzFloat *elm, int num, AzFloat border, 
                                     AzFloat *deriv_elm) /* must not be NULL */ {
    int i; 
    for (i = azc_thno; i < num; i +=azc_thnum) {
      if (elm[i] < -border) {
        elm[i] = -border; 
        deriv_elm[i] = 0; 
      } 
      else if (elm[i] > border) {
        elm[i] = border; 
        deriv_elm[i] = 0; 
      } 
    }
  }  
  void azccall_truncate(AzFloat *elm, int num, AzFloat border, 
                        AzFloat *deriv_elm) /* may be NULL */ {
    int bb, tt; 
    azc_config(num, bb, tt, "azccall_truncate"); 
    if (deriv_elm == NULL) azc_kernel(azc_truncate,bb,tt)(elm, num, border); 
    else                   azc_kernel(azc_truncate_deriv,bb,tt)(elm, num, border, deriv_elm); 
    chk_err("azccall_truncate", bb, tt); 
  }  

  /*******           For convolutional layers             *******/
  /*------------------------------------------------------------*/
  /*---              filtering/unfiltering                   ---*/
  /*------------------------------------------------------------*/
  __global__ void azc_add_with_map(int num, const azcparam_add_with_map p) 
  {
    int ex;   
    for (ex = azc_thno; ex < num; ex += azc_thnum) {
      int index = ex;  
      int row = index % p.row_num; 
      index = index / p.row_num; 
      int dx = index / p.width2; 
      int col2 = index % p.width2; 
      if (dx >= p.data_num) continue;  
    
      int base1 = dx*p.width1; 
      int base2 = dx*p.width2; 
  
      const int *col1 = _column(col2, p.a2to1, p.nummax); 
      AzFloat *e2 = _column(base2+col2, p.elm2, p.row_num); 
      int ix; 
      for (ix = 0; ix < p.nummax; ++ix) {
        if (col1[ix] == p.stopper) break; 
        const AzFloat *e1 = _column(base1+col1[ix], p.elm1, p.row_num); 
        e2[row] += e1[row];     
      }
    }
  }

  /*------------------------------------------------------------*/
  void azccall_add_with_map(const azcparam_add_with_map p) {
    int num = p.data_num * p.width2 * p.row_num; 
    int bb, tt; 
    azc_config(num, bb, tt, "azccall_add_with_map"); 
    azc_kernel(azc_add_with_map,bb,tt)(num, p); 
    chk_err("azccall_add_with_map", bb, tt); 
  }
 
  /*----------------------------------------------------------------------
   *   add_with_map_var: Added on 2/14/2017 for speed-up. Old "fastVar=2".
   *   - a2to1 gives a template for the largest data.
   *     Watch out for col1's pointing outside.  
   *   - This is almost as fast as and more flexible than 
   *     the "no-template" version. 
   *---------------------------------------------------------------------*/   
  __global__ void azc_add_with_map_var(int num, const azcparam_var p) {
    for (int ex = azc_thno; ex < num; ex += azc_thnum) {
      int row = ex % p.row_num; 
      int c2 = ex / p.row_num; 

      int dx = p.c2dx[c2/p.c2dx_denomi]; 
      
      int base1 = p.pos1[dx], num1 = p.pos1[dx+1] - base1;     
      int col2 = c2 - p.pos2[dx]; 

      const int *col1_arr = _column(col2, p.col2to1_ptr, p.nummax); 
      AzFloat *e2 = _column(c2, p.elm2, p.row_num); 

      for (int ix = 0; ix < p.nummax; ++ix) {
        int col1 = col1_arr[ix]; 
        if (col1 == p.stopper) break; 
        if (col1 >= num1) continue; 
        const AzFloat *e1 = _column(base1+col1, p.elm1, p.row_num); 
        e2[row] += e1[row];     
      }
    }
  }  
  /*------------------------------------------------------------*/
  void azccall_add_with_map_var(const azcparam_var p, int cnum2) {
    AzX::throw_if((cnum2 <= 0), "azccall_add_with_map_var", "cnum2 must be positive"); 
    int num = cnum2 * p.row_num; 
    int bb, tt; azc_config(num, bb, tt, "azccall_add_with_map_var");    
    azc_kernel(azc_add_with_map_var,bb,tt)(num, p);       
    chk_err("azccall_add_with_map_var", bb, tt); 
  }   

  /*******                   Pooling                      *******/  
  /*------------------------------------------------------------*/
  /*---                  average pooling                     ---*/
  /*------------------------------------------------------------*/
  __global__ void azc_pooling_avg(int num, const azcparam_pooling_avg p) 
  {
    int ex;   
    for (ex = azc_thno; ex < num; ex += azc_thnum) {
      int index = ex; 
      int row = index % p.row_num; 
      index = index / p.row_num;   
      int dx = index / p.width2; 
      int col2 = index % p.width2; 
      if (dx >= p.data_num) continue; 

      int base1 = dx*p.width1; 
      int base2 = dx*p.width2; 

      const int *col1 = _column(col2, p.col1_ptr, p.col1_nummax); 
      AzFloat *e2 = _column(base2+col2, p.elm2, p.row_num); 
      int ix; 
      for (ix = 0; ix < p.col1_nummax; ++ix) {
        if (col1[ix] == p.stopper) break; 
        const AzFloat *e1 = _column(base1+col1[ix], p.elm1, p.row_num); 
        e2[row] += e1[row];  
      }
      int col1_num = ix; 
      if (col1_num != 0) e2[row] /= (AzFloat)col1_num; 
    }
  }
  /*------------------------------------------------------------*/
  void azccall_pooling_avg(const azcparam_pooling_avg p) {
    int num = p.data_num * p.width2 * p.row_num; 
    int bb, tt; 
    azc_config(num, bb, tt, "azccall_pooling_avg"); 
    azc_kernel(azc_pooling_avg,bb,tt)(num, p); 
    chk_err("azccall_pooling_avg", bb, tt);         
  }              
  
  /*------------------------------------------------------------*/
  /*---                  average unpooling                   ---*/
  /*------------------------------------------------------------*/
  __global__ void azc_unpooling_avg(int num, const azcparam_unpooling_avg p) {
    int ex;   
    for (ex = azc_thno; ex < num; ex += azc_thnum) {
      int index = ex;     
      int row = index % p.row_num; 
      index = index / p.row_num;         
      int dx = index / p.width1; 
      int col1 = index % p.width1; 
      if (dx >= p.data_num) continue; 

      int base1 = dx*p.width1; 
      int base2 = dx*p.width2;   

      const int *col2 = _column(col1, p.col2_ptr, p.col2_nummax); 
      int ix; 
      for (ix = 0; ix < p.col2_nummax; ++ix) {
        if (col2[ix] == p.stopper) break;       
        AzFloat *e1 = _column(base1+col1, p.elm1, p.row_num); 
        const AzFloat *e2 = _column(base2+col2[ix], p.elm2, p.row_num); 
        AzFloat denomi = (AzFloat)p.col2_to_num[col2[ix]]; 
        if (denomi != 0) e1[row] += e2[row]/denomi; 
      }
    }
  }

  /*------------------------------------------------------------*/
  void azccall_unpooling_avg(const azcparam_unpooling_avg p) {
    int num = p.data_num * p.width1 * p.row_num; 
    int bb, tt; 
    azc_config(num, bb, tt, "azccall_unpooling_avg"); 
    azc_kernel(azc_unpooling_avg,bb,tt)(num, p); 
    chk_err("azccall_unpooling_avg", bb, tt);         
  }

  /*------------------------------------------------------------*/
  /*---                     l2 pooling                       ---*/
  /*------------------------------------------------------------*/
  __global__ void azc_pooling_l2(int num, const azcparam_pooling_l2 p) {
    int ex;   
    for (ex = azc_thno; ex < num; ex += azc_thnum) {
      int index = ex;   
      int row = index % p.row_num; 
      index = index / p.row_num; 
      int dx = index / p.width2; 
      int col2 = index % p.width2; 
      if (dx >= p.data_num) continue; 

      int base1 = dx*p.width1; 
      int base2 = dx*p.width2; 

      const int *col1 = _column(col2, p.col1_ptr, p.col1_nummax); 
      AzFloat *e2 = _column(base2+col2, p.elm2, p.row_num); 
      int ix; 
      for (ix = 0; ix < p.col1_nummax; ++ix) {
        if (col1[ix] == p.stopper) break; 
        const AzFloat *e1 = _column(base1+col1[ix], p.elm1, p.row_num); 
        e2[row] += e1[row]*e1[row]; 
      }
      e2[row] = sqrt(e2[row]); 
    }
  }

  /*------------------------------------------------------------*/
  void azccall_pooling_l2(const azcparam_pooling_l2 p) {
    int num = p.data_num * p.width2 * p.row_num; 
    int bb, tt; 
    azc_config(num, bb, tt, "azccall_pooling_l2");  
    azc_kernel(azc_pooling_l2,bb,tt)(num, p); 
    chk_err("azccall_pooling_l2", bb, tt);         
  }
 
  /*------------------------------------------------------------*/
  /*---                    l2 unpooling                      ---*/
  /*------------------------------------------------------------*/
  __global__ void azc_unpooling_l2(int num, const azcparam_unpooling_l2 p) {
    int ex;   
    for (ex = azc_thno; ex < num; ex += azc_thnum) {
      int index = ex; 
      int row = index % p.row_num; 
      index = index / p.row_num;    
      int dx = index / p.width1; 
      int col1 = index % p.width1; 
      if (dx >= p.data_num) continue; 

      int base1 = dx*p.width1; 
      int base2 = dx*p.width2;   

      AzFloat *e1 = _column(base1+col1, p.elm1, p.row_num); 
      const AzFloat *org_e1 = _column(base1+col1, p.org_elm1, p.row_num); 
      const int *col2 = _column(col1, p.col2_ptr, p.col2_nummax); 
      int ix; 
      for (ix = 0; ix < p.col2_nummax; ++ix) {
        if (col2[ix] == p.stopper) break;       
        const AzFloat *e2 = _column(base2+col2[ix], p.elm2, p.row_num); 
        const AzFloat *org_e2 = _column(base2+col2[ix], p.org_elm2, p.row_num); 
        if (org_e2[row] != 0) e1[row] += (e2[row] * org_e1[row] / org_e2[row]); 
      }
    }
  }

  /*------------------------------------------------------------*/
  void azccall_unpooling_l2(const azcparam_unpooling_l2 p) {
    int num = p.data_num * p.width1 * p.row_num; 
    int bb, tt; 
    azc_config(num, bb, tt, "azccall_unpooling_l2"); 
    azc_kernel(azc_unpooling_l2,bb,tt)(num, p); 
    chk_err("azccall_unpooling_l2", bb, tt);         
  }
  
  /*------------------------------------------------------------*/
  /*---                    max pooling                       ---*/
  /*------------------------------------------------------------*/
  __global__ void azc_pooling_max(int num, const azcparam_pooling_max p) {
    int ex;   
    for (ex = azc_thno; ex < num; ex += azc_thnum) {
      int index = ex; 
      int row = index % p.row_num; 
      index = index / p.row_num;         
      int dx = index / p.width2; 
      int col2 = index % p.width2; 
      if (dx >= p.data_num) continue; 
  
      int base1 = dx*p.width1; 
      int base2 = dx*p.width2; 

      const int *col1 = _column(col2, p.col1_ptr, p.col1_nummax); 
      int *chosen = (p.chosen_ptr != NULL) ? _column(col2+base2, p.chosen_ptr, p.row_num) : NULL; 
      AzFloat *max_elm = _column(col2+base2, p.elm2, p.row_num); 
      int ix; 
      for (ix = 0; ix < p.col1_nummax; ++ix) {
        if (col1[ix] == p.stopper) break; 
        const AzFloat *e1 = _column(col1[ix]+base1, p.elm1, p.row_num); 
        if (ix == 0 || e1[row] > max_elm[row]) {
          max_elm[row] = e1[row]; 
          if (chosen != NULL) chosen[row] = col1[ix]; 
        }
      }    
    }                         
  }
  /*------------------------------------------------------------*/
  void azccall_pooling_max(const azcparam_pooling_max p) {
    int num = p.data_num * p.width2 * p.row_num; 
    int bb, tt; 
    azc_config(num, bb, tt, "azccall_pooling_max"); 
    azc_kernel(azc_pooling_max,bb,tt)(num, p); 
    chk_err("azccall_pooling_max", bb, tt);         
  }
  /*------------------------------------------------------------*/  
  __global__ void azc_pooling_max_up2(int num, const azcparam_pooling_max p, const int *_chosen) {
    int ex;   
    for (ex = azc_thno; ex < num; ex += azc_thnum) {
      int index = ex; 
      int row = index % p.row_num; 
      index = index / p.row_num;         
      int dx = index / p.width2; 
      int col2 = index % p.width2; 
      if (dx >= p.data_num) continue; 
  
      int base1 = dx*p.width1; 
      int base2 = dx*p.width2; 

      const int *chosen = _column(col2+base2, _chosen, p.row_num); 
      AzFloat *out_elm2 = _column(col2+base2, p.elm2, p.row_num); 
      int chosen_col1 = chosen[row];       
      const AzFloat *e1 = _column(chosen_col1+base1, p.elm1, p.row_num); 
      out_elm2[row] = e1[row]; 
    }                         
  }
  /*------------------------------------------------------------*/
  void azccall_pooling_max_up2(const azcparam_pooling_max p, const int *_chosen) {
    int num = p.data_num * p.width2 * p.row_num; 
    int bb, tt; azc_config(num, bb, tt, "azccall_pooling_max_up2"); 
    azc_kernel(azc_pooling_max_up2,bb,tt)(num, p, _chosen); 
    chk_err("azccall_pooling_max_up2", bb, tt);         
  }  
  
  /*------------------------------------------------------------*/  
  /*---                   max unpooling                      ---*/
  /*------------------------------------------------------------*/
  /* Note: assume overlapping pooling */
  /* thread: portions of rows of one data point  */
  __global__ void azc_unpooling_max(int num, const azcparam_unpooling_max p) {
    int ex;   
    for (ex = azc_thno; ex < num; ex += azc_thnum) {
      int index = ex;   
      int row = index % p.row_num; 
      index = index / p.row_num;    
      int dx = index % p.data_num; 
      if (dx >= p.data_num) continue; 
  
      int base1 = dx*p.width1; 
      int base2 = dx*p.width2; 
    
      int col2; 
      for (col2 = 0; col2 < p.width2; ++col2) {
        const AzFloat *e2 = _column(col2+base2, p.elm2, p.row_num); 
        const int *chosen = _column(col2+base2, p.ptr_chosen, p.row_num); 
        int col1 = chosen[row]; 
        if (col1 >= 0) { /* 3/11/2014: for variable-length pooling */
          (_column(col1+base1, p.elm1, p.row_num))[row] += e2[row]; 
        }
      }
    }
  }
  /*------------------------------------------------------------*/
  void azccall_unpooling_max(const azcparam_unpooling_max p) {
    int num = p.data_num * p.row_num; 
    int bb, tt; 
    azc_config(num, bb, tt, "azccall_unpooling_max"); 
    azc_kernel(azc_unpooling_max,bb,tt)(num, p); 
    chk_err("azccall_unpooling_max", bb, tt);         
  }
  
  /*------------------------------------------------------------*/  
  __global__ void azc_unpooling_max2(int num, const azcparam_unpooling_max p) {
    for (int ex = azc_thno; ex < num; ex += azc_thnum) {
      int index = ex; 
      int row = index % p.row_num; 
      index = index / p.row_num;         
      int dx = index / p.width1; 
      int col1 = index % p.width1; 
      if (dx >= p.data_num) return; 

      int base1 = dx*p.width1; int base2 = dx*p.width2;   

      AzFloat *e1 = _column(base1+col1, p.elm1, p.row_num); 
      const int *col2 = _column(col1, p.col1to2_ptr, p.nummax); 
      for (int ix = 0; ix < p.nummax; ++ix) {
        if (col2[ix] == p.stopper) break;       
        const int *chosen = _column(base2+col2[ix], p.ptr_chosen, p.row_num); 
        if (chosen[row] == col1) {
          const AzFloat *e2 = _column(base2+col2[ix], p.elm2, p.row_num); 
          e1[row] += e2[row]; 
        }
      }  
    }    
  }  
  /*------------------------------------------------------------*/
  void azccall_unpooling_max2(const azcparam_unpooling_max p) {
    int num = p.data_num * p.width1 * p.row_num; 
    int bb, tt; azc_config(num, bb, tt, "azccall_unpooling_max2"); 
    azc_kernel(azc_unpooling_max2,bb,tt)(num, p); 
    chk_err("azccall_unpooling_max2", bb, tt);         
  }   


  /***  To speed-up pooling with variable-sized input/output  ***/
   /*------------------------------------------------------------*/
  __global__ void azc_pooling_max_var(int num, const azcparam_var p, int *_chosen) {
    for (int ex = azc_thno; ex < num; ex += azc_thnum) {
      int row = ex % p.row_num, c2 = ex / p.row_num, dx = p.c2dx[c2/p.c2dx_denomi]; 
      int base1 = p.pos1[dx], num1 = p.pos1[dx+1] - base1; 
      int col2 = c2 - p.pos2[dx];     
      
      const int *col1_arr = _column(col2, p.col2to1_ptr, p.nummax); 
      
      int *chosen = (_chosen != NULL) ? _column(c2, _chosen, p.row_num) : NULL; 
      AzFloat *max_elm = _column(c2, p.elm2, p.row_num); 
      for (int ix = 0; ix < p.nummax; ++ix) {
        int col1 = col1_arr[ix]; 
        if (col1 == p.stopper) break; 
        if (col1 >= num1) continue;                
        const AzFloat *e1 = _column(col1+base1, p.elm1, p.row_num); 
        if (ix == 0 || e1[row] > max_elm[row]) {
          max_elm[row] = e1[row]; 
          if (chosen != NULL) chosen[row] = col1; 
        }
      }    
    }                         
  }
  /*------------------------------------------------------------*/
  void azccall_pooling_max_var(const azcparam_var p, int cnum2, int *_chosen) {
    int num = cnum2 * p.row_num; 
    int bb, tt; azc_config(num, bb, tt, "azccall_pooling_max_var"); 
    azc_kernel(azc_pooling_max_var,bb,tt)(num, p, _chosen); 
    chk_err("azccall_pooling_max_var", bb, tt);         
  }
  /*------------------------------------------------------------*/
  __global__ void azc_pooling_max_var_up2(int num, const azcparam_var p, const int *_chosen) {
    for (int ex = azc_thno; ex < num; ex += azc_thnum) {
      int row = ex % p.row_num, c2 = ex / p.row_num, dx = p.c2dx[c2/p.c2dx_denomi]; 
      int base1 = p.pos1[dx]; 
      int chosen_col1 = _entry(row, c2, _chosen, p.row_num);  
      const AzFloat *e1 = _column(chosen_col1+base1, p.elm1, p.row_num); 
      AzFloat *out_elm2 = _column(c2, p.elm2, p.row_num);       
      out_elm2[row] = e1[row];    
    }                         
  }
  /*------------------------------------------------------------*/
  void azccall_pooling_max_var_up2(const azcparam_var p, int cnum2, const int *_chosen) {
    int num = cnum2 * p.row_num; 
    int bb, tt; azc_config(num, bb, tt, "azccall_pooling_max_var_up2"); 
    azc_kernel(azc_pooling_max_var_up2,bb,tt)(num, p, _chosen); 
    chk_err("azccall_pooling_max_var_up2", bb, tt);         
  }   
  
  /*------------------------------------------------------------*/
  __global__ void azc_pooling_avg_var(int num, const azcparam_var p, int sz) {
    for (int ex = azc_thno; ex < num; ex += azc_thnum) {
      int row = ex % p.row_num, c2 = ex / p.row_num, dx = p.c2dx[c2/p.c2dx_denomi]; 
      int base1 = p.pos1[dx], num1 = p.pos1[dx+1] - base1; 
      int col2 = c2 - p.pos2[dx]; 

      const int *col1_arr = _column(col2, p.col2to1_ptr, p.nummax); 
 
      AzFloat *ptr2 = _column(c2, p.elm2, p.row_num) + row; 
      for (int ix = 0; ix < p.nummax; ++ix) {
        int col1 = col1_arr[ix]; 
        if (col1 == p.stopper) break;         
        if (col1 >= num1) continue; 
        *ptr2 += _entry(row, col1+base1, p.elm1, p.row_num);  
      }    
      *ptr2 /= (AzFloat)sz; 
    }                         
  }
  /*------------------------------------------------------------*/
  void azccall_pooling_avg_var(const azcparam_var p, int cnum2, int sz) {
    int num = cnum2 * p.row_num; 
    int bb, tt; azc_config(num, bb, tt, "azccall_pooling_avg_var"); 
    azc_kernel(azc_pooling_avg_var,bb,tt)(num, p, sz); 
    chk_err("azccall_pooling_avg_var", bb, tt);         
  }
   
  /*------------------------------------------------------------*/
  __global__ void azc_unpooling_max_var(int num, const azcparam_var p, const int *_chosen) {
    for (int ex = azc_thno; ex < num; ex += azc_thnum) { 
      int row = ex%p.row_num, c2 = ex/p.row_num, dx = p.c2dx[c2/p.c2dx_denomi];       
      int col2 = c2 - p.pos2[dx]; 
      int base1 = p.pos1[dx], num1 = p.pos1[dx+1] - base1;
      
      AzFloat *e2 = _column(c2, p.elm2, p.row_num); 
      const int *col1_arr = _column(col2, p.col2to1_ptr, p.nummax); 
      for (int ix = 0; ix < p.nummax; ++ix) {
        int col1 = col1_arr[ix]; 
        if (col1 == p.stopper) break;            
        if (col1 >= num1) continue; 
     
        if (col1 < 0) {
          if (row == 0) 
          printf("azc_unpooling_max_var col1 < 0, col1=%d, dx=%d, num1=%d, azc_thno=%d, ex=%d, row=%d, c2=%d, col2=%d\n", 
                 col1, dx, num1, azc_thno, ex, row, c2, col2); 
          continue; 
        }
       
        const int *chosen = _column(base1+col1, _chosen, p.row_num); 
        if (chosen[row] == col2) {
          const AzFloat *e1 = _column(base1+col1, p.elm1, p.row_num);            
          e2[row] += e1[row]; 
        }
      }  
    }    
  }
  /*------------------------------------------------------------*/
  void azccall_unpooling_max_var(const azcparam_var p, int cnum2, const int *_chosen) {
    int num = cnum2 * p.row_num; 
    int bb, tt; azc_config(num, bb, tt, "azccall_unpooling_max_var");  
    azc_kernel(azc_unpooling_max_var,bb,tt)(num, p, _chosen); 
    chk_err("azccall_unpooling_max_var", bb, tt);         
  }   
   
  /*------------------------------------------------------------*/
  __global__ void azc_unpooling_avg_var(int num, const azcparam_var p, int sz) {
    for (int ex = azc_thno; ex < num; ex += azc_thnum) { 
      int row = ex % p.row_num, c2 = ex / p.row_num, dx = p.c2dx[c2/p.c2dx_denomi];       
      int col2 = c2 - p.pos2[dx]; 
      int base1 = p.pos1[dx], num1 = p.pos1[dx+1] - base1;
      
      AzFloat *ptr2 = _column(c2, p.elm2, p.row_num) + row; 
      const int *col1_arr = _column(col2, p.col2to1_ptr, p.nummax); 
      for (int ix = 0; ix < p.nummax; ++ix) {
        int col1 = col1_arr[ix]; 
        if (col1 == p.stopper) break;       
        if (col1 >= num1) continue; 
        *ptr2 += (_entry(row, base1+col1, p.elm1, p.row_num)/(AzFloat)sz); 
      }  
    }    
  }
  /*------------------------------------------------------------*/
  void azccall_unpooling_avg_var(const azcparam_var p, int cnum2, int sz) {
    int num = cnum2 * p.row_num; 
    int bb, tt; azc_config(num, bb, tt, "azccall_unpooling_avg_var"); 
    azc_kernel(azc_unpooling_avg_var,bb,tt)(num, p, sz); 
    chk_err("azccall_unpooling_avg_var", bb, tt);         
  }   
  
  /*******                    Misc                        *******/    
  /*------------------------------------------------------------*/    
  /*---      rearrange (for locally-connected weights)       ---*/
  /*------------------------------------------------------------*/    
  __global__ void azc_rearrange(int num, const azcparam_rearrange p) {
    int ex;   
    for (ex = azc_thno; ex < num; ex += azc_thnum) {
      int index = ex;   
      int row = index % p.rnum; 
      index = index / p.rnum;         
      int dx = index / p.loc_num; 
      int loc = index % p.loc_num; 
      if (dx >= p.d_num) continue; 

      const AzFloat *e1 = _column(p.loc_num*dx + loc, p.elm1, p.rnum); 
      AzFloat *e2 = _column(p.d_num*loc + dx, p.elm2, p.rnum); 
      e2[row] = e1[row];    
    }
  }

  /*------------------------------------------------------------*/
  void azccall_rearrange(const azcparam_rearrange p) {
    int num = p.d_num * p.loc_num * p.rnum; 
    int bb, tt; 
    azc_config(num, bb, tt, "azccall_rearrange"); 
    azc_kernel(azc_rearrange,bb,tt)(num, p); 
    chk_err("azccall_rearrange", bb, tt);         
  }    

  /*------------------------------------------------------------*/  
  /*------------------------------------------------------------*/      
  __global__ void azc_undo_rearrange(int num, const azcparam_undo_rearrange p) {
    int ex;   
    for (ex = azc_thno; ex < num; ex += azc_thnum) {
      int index = ex;   
      int row = index % p.rnum; 
      index = index / p.rnum;         
      int dx = index / p.loc_num; 
      int loc = index % p.loc_num; 
      if (dx >= p.d_num) continue; 

      AzFloat *e1 = _column(p.loc_num*dx + loc, p.elm1, p.rnum); 
      const AzFloat *e2 = _column(p.d_num*loc + dx, p.elm2, p.rnum);     
      e1[row] = e2[row]; 
    }
  }

  /*------------------------------------------------------------*/
  void azccall_undo_rearrange(const azcparam_undo_rearrange p) {
    int num = p.d_num * p.loc_num * p.rnum; 
    int bb, tt; 
    azc_config(num, bb, tt, "azccall_undo_rearrange"); 
    azc_kernel(azc_undo_rearrange,bb,tt)(num, p); 
    chk_err("azccall_undo_rearrange", bb, tt);         
  }   

  /*------------------------------------------------------------*/
  /*-- local response normalization across neurons  (cmrnorm) --*/
  /*------------------------------------------------------------*/
  __global__ void azc_resnorm_crossmap(int num, const azcparam_resnorm_crossmap p) {
    int ex;   
    for (ex = azc_thno; ex < num; ex += azc_thnum) {
      int index = ex;   
      int row = index % p.rnum; 
      int col = index / p.rnum; 
      if (col >= p.cnum) continue; 
  
      int sz, rr0, rr1; 
      if (p.size < p.rnum) {
        int halfsize = p.size / 2; 
        sz = halfsize*2 + 1; 
        sz = 1; /* to match with cuda-convnet */
        rr0 = row - halfsize;
        rr1 = row + halfsize; 
      }
      else {
        sz = 1; 
        rr0 = 0; 
        rr1 = p.rnum-1; 
      }
    
      int base = col*p.rnum; 
      const AzFloat *input = p.elm + base; 
      AzFloat *oneplussqavg = p.elm_oneplussqavg + base; 
      AzFloat *normalized = p.elm_normalized + base;   

      AzFloat sqsum = 0; 
      int rr; 
      for (rr = rr0; rr <= rr1; ++rr) {
        int myrr = (rr+p.rnum) % p.rnum; 
        sqsum += input[myrr]*input[myrr]; 
      }
      oneplussqavg[row] = p.one+p.alpha*sqsum/(double)sz; 
      normalized[row] *= pow(oneplussqavg[row], -p.beta); 
    }
  }
  
  /*------------------------------------------------------------*/
  void azccall_resnorm_crossmap(const azcparam_resnorm_crossmap p) {
    int num = p.cnum * p.rnum; 
    int bb, tt; 
    azc_config(num, bb, tt, "azccall_resnorm_crossmap");  
    azc_kernel(azc_resnorm_crossmap,bb,tt)(num, p); 
    chk_err("azccall_resnorm_crossmap", bb, tt);         
  }
  
  /*------------------------------------------------------------*/
  /*-- local response normalization across channels (cmrnorm) --*/
  /*------------------------------------------------------------*/
  __global__ void azc_resnorm_crossmap_all(int num, const azcparam_resnorm_crossmap p, 
                  const AzFloat *col_sqsum) {
    for (int ex = azc_thno; ex < num; ex += azc_thnum) {
      int index = ex;   
      int row = index % p.rnum; 
      int col = index / p.rnum; 
      if (col >= p.cnum) continue; 

      int base = col*p.rnum; 
      AzFloat *oneplussqavg = p.elm_oneplussqavg + base; 
      AzFloat *normalized = p.elm_normalized + base;   
      
      oneplussqavg[row] = p.one+p.alpha*col_sqsum[col]; 
      normalized[row] *= pow(oneplussqavg[row], -p.beta); 
    }
  }
  
  /*------------------------------------------------------------*/
  void azccall_resnorm_crossmap_all(const azcparam_resnorm_crossmap p, const AzFloat *col_sqsum) {
    int num = p.cnum * p.rnum; 
    int bb, tt; 
    azc_config(num, bb, tt, "azccall_resnorm_crossmap_all");  
    azc_kernel(azc_resnorm_crossmap_all,bb,tt)(num, p, col_sqsum); 
    chk_err("azccall_resnorm_crossmap_all", bb, tt);         
  }  
  
  /*-------------------------------------------------------------*/  
  /*---  prep for undoing response normalization (cross map)  ---*/
  /*-------------------------------------------------------------*/
  /* tmp <- g_k * (-2 alpha beta)/N_k * v_k(1 + alpha/N sum_i v_i^2)^{-beta-1} */
  /*     =  (-2 alpha beta f_k g_k)/(N_k d_k) */
  /*  d_k := 1 + alpha/N_k sum_i v_i^2 */
  /*  f_k := v_k d_k^{-beta} */
  __global__ void azc_prep_unresnorm_crossmap(int num, const azcparam_prep_unresnorm_crossmap p) {
    int ex;   
    for (ex = azc_thno; ex < num; ex += azc_thnum) {
      int index = ex;   
      int row = index % p.rnum; 
      int col = index / p.rnum; 
      if (col >= p.cnum) continue; 

      int sz = 1; 
      int base = p.rnum*col; 
      const AzFloat *grad = p.elm_grad + base; 
      const AzFloat *aft = p.elm_aft + base; 
      const AzFloat *oneplussqavg = p.elm_oneplussqavg + base; 
      AzFloat *tmp = p.elm_tmp + base; 

      tmp[row] = (-2*p.alpha*p.beta * aft[row]*grad[row]) / (oneplussqavg[row] * (double)sz); 
    }
  }

  /*------------------------------------------------------------*/
  void azccall_prep_unresnorm_crossmap(const azcparam_prep_unresnorm_crossmap p) {
    int num = p.cnum * p.rnum; 
    int bb, tt; 
    azc_config(num, bb, tt, "azccall_prep_unresnorm_crossmap"); 
    azc_kernel(azc_prep_unresnorm_crossmap,bb,tt)(num, p); 
    chk_err("azccall_prep_unresnorm_crossmap", bb, tt);         
  }  

  /*-------------------------------------------------------------*/  
  /*---      undo response normalization (cross map)        ---*/
  /*-------------------------------------------------------------*/
  /* v_j sum_k (-2 alpha beta f_k g_k)/(N_k d_k)  +  (f_j g_j) / v_j */
  /*------------------------------------------------------------*/
  __global__ void azc_unresnorm_crossmap(int num, const azcparam_unresnorm_crossmap p) {
    int ex;   
    for (ex = azc_thno; ex < num; ex += azc_thnum) {
      int index = ex; 
      int row = index % p.rnum; 
      int col = index / p.rnum; 
      if (col >= p.cnum) continue; 

      int rr0, rr1; 
      if (p.size < p.rnum) {
        int halfsize = p.size / 2; 
        rr0 = row - halfsize;
        rr1 = row + halfsize; 
      }
      else {
        rr0 = 0; 
        rr1 = p.rnum-1; 
      }
    
      int base = col*p.rnum; 
      const AzFloat *tmp = p.elm_tmp + base;  
      const AzFloat *bef = p.elm_bef + base;  
      const AzFloat *oneplussqavg = p.elm_oneplussqavg + base;       
      const AzFloat *grad = p.elm_grad + base;  
      AzFloat *out = p.elm_out + base; 

      AzFloat val = 0; 
      if (bef[row] != 0) {
        int rr; 
        for (rr = rr0; rr <= rr1; ++rr) { /* neighbor relationship is mutual */
          int myrr = (rr+p.rnum)%p.rnum; 
          val += tmp[myrr]; 
        }
      }
      out[row] = bef[row]*val + grad[row]*pow(oneplussqavg[row],-p.beta); 
    }
  }

  /*------------------------------------------------------------*/
  void azccall_unresnorm_crossmap(const azcparam_unresnorm_crossmap p) {
    int num = p.cnum * p.rnum; 
    int bb, tt; 
    azc_config(num, bb, tt, "azccall_unresnorm_crossmap");  
    azc_kernel(azc_unresnorm_crossmap,bb,tt)(num, p); 
    chk_err("azccall_unresnorm_crossmap", bb, tt);         
  } 
  
  /*------------------------------------------------------------*/
  __global__ void azc_unresnorm_crossmap_all(int num, const azcparam_unresnorm_crossmap p, const AzFloat *tmp_colSum) {
    int ex;   
    for (ex = azc_thno; ex < num; ex += azc_thnum) {
      int index = ex; 
      int row = index % p.rnum; 
      int col = index / p.rnum; 
      if (col >= p.cnum) continue; 

      int base = col*p.rnum; 
      const AzFloat *bef = p.elm_bef + base;  
      const AzFloat *oneplussqavg = p.elm_oneplussqavg + base;       
      const AzFloat *grad = p.elm_grad + base;  
      AzFloat *out = p.elm_out + base; 

      out[row] = bef[row]*tmp_colSum[col] + grad[row]*pow(oneplussqavg[row],-p.beta); 
    }
  }

  /*------------------------------------------------------------*/
  void azccall_unresnorm_crossmap_all(const azcparam_unresnorm_crossmap p, const AzFloat *tmp_colSum) {
    int num = p.cnum * p.rnum; 
    int bb, tt; 
    azc_config(num, bb, tt, "azccall_unresnorm_crossmap_all");  
    azc_kernel(azc_unresnorm_crossmap_all,bb,tt)(num, p, tmp_colSum); 
    chk_err("azccall_unresnorm_crossmap_all", bb, tt);         
  } 
  
  /*------------------------------------------------------------*/
  /*---            response normalization (local)            ---*/
  /*------------------------------------------------------------*/  
  __global__ void azc_resnorm_local(int num, const azcparam_resnorm_local p) {
    int ex;   
    for (ex = azc_thno; ex < num; ex += azc_thnum) {
      int index = ex;  
      int row = index % p.rnum; 
      index = index / p.rnum;    
      int dx = index / p.cnum; 
      int col = index % p.cnum; 
      if (dx >= p.data_num) continue;  

      int base = dx*p.cnum; 
    
      /*---  compute  1 + alpha avg_i v_i^2  ---*/
      const int *neigh = _column(col, p.neighbors, p.nummax); 
      AzFloat *oneplussqavg = _column(base+col, p.elm_oneplussqavg, p.rnum); 
      AzFloat *normalized = _column(base+col, p.elm_normalized, p.rnum); 
      int ix; 
      for (ix = 0; ix < p.nummax; ++ix) {
        if (neigh[ix] == p.stopper) break; 
        const AzFloat *neigh_val = _column(base+neigh[ix], p.elm, p.rnum); 
        oneplussqavg[row] += neigh_val[row]*neigh_val[row]; 
      }
      int sz = p.neigh_sz[col]; 
  
      sz = 1; /* to match with cuda-convnet */
  
      AzFloat coeff = 1; 
      if (sz != 0) {
        coeff = p.alpha / (AzFloat)sz; 
      }
      oneplussqavg[row] *= coeff; 
      oneplussqavg[row] += 1; 
      /*---  multiply (1 + alpha avg_i v_i^2)^{-beta}  ---*/
      normalized[row] *= pow(oneplussqavg[row], -p.beta); 
    }
  }

  /*------------------------------------------------------------*/
  void azccall_resnorm_local(const azcparam_resnorm_local p) {
    int num = p.data_num * p.cnum * p.rnum; 
    int bb, tt; 
    azc_config(num, bb, tt, "azccall_resnorm_local"); 
    azc_kernel(azc_resnorm_local,bb,tt)(num, p); 
    chk_err("azccall_resnorm_local", bb, tt);         
  } 

  /*------------------------------------------------------------*/  
  /*---   prep for undoing response normalization (local)    ---*/
  /*------------------------------------------------------------*/
  /* tmp <- g_k * (-2 alpha beta)/N_k * v_k(1 + alpha/N sum_i v_i^2)^{-beta-1} */
  /*     =  (-2 alpha beta f_k g_k)/(N_k d_k)                   */
  /*  d_k := 1 + alpha/N_k sum_i v_i^2                          */
  /*  f_k := v_k d_k^{-beta}                                    */
  /*------------------------------------------------------------*/
  __global__ void azc_prep_unresnorm_local(int num, const azcparam_prep_unresnorm_local p) { 
    int ex;   
    for (ex = azc_thno; ex < num; ex += azc_thnum) {
      int index = ex; 
      int row = index % p.rnum; 
      index = index / p.rnum;    
      int dx = index / p.cnum; 
      int col = index % p.cnum; 
      if (dx >= p.data_num) continue; 

      int base0 = dx*p.rnum*p.cnum; 
      int base = base0 + p.rnum*col; 
      const AzFloat *grad = p.elm_grad + base; 
      const AzFloat *aft = p.elm_aft + base; 
      const AzFloat *oneplussqavg = p.elm_oneplussqavg + base; 
      AzFloat *tmp = p.elm_tmp + base; 
      int sz = p.neigh_sz[col]; 

      sz = 1; /* to match with cuda-convnet */

      tmp[row] = (-2*p.alpha*p.beta * aft[row]*grad[row]) / (oneplussqavg[row] * (AzFloat)sz); 
    }
  }

  /*------------------------------------------------------------*/
  void azccall_prep_unresnorm_local(const azcparam_prep_unresnorm_local p) {
    int num = p.data_num * p.cnum * p.rnum; 
    int bb, tt; 
    azc_config(num, bb, tt, "azccall_prep_unresnorm_local"); 
    azc_kernel(azc_prep_unresnorm_local,bb,tt)(num, p);   
    chk_err("azccall_prep_unresnorm_local", bb, tt);         
  }    

  /*------------------------------------------------------------*/  
  /*---          undo response normalization (local)         ---*/  
  /* v_j sum_k (-2 alpha beta f_k g_k)/(N_k d_k)  +  (f_j g_j) / v_j */
  /*------------------------------------------------------------*/
  __global__ void azc_unresnorm_local(int num, const azcparam_unresnorm_local p) { 
    int ex;   
    for (ex = azc_thno; ex < num; ex += azc_thnum) {
      int index = ex; 
      int row = index % p.rnum; 
      index = index / p.rnum;    
      int dx = index / p.cnum; 
      int col = index % p.cnum; 
      if (dx >= p.data_num) continue; 

      int base0 = dx*p.rnum*p.cnum; 
      const AzFloat *tmp = p.elm_tmp + base0;  

      int base = base0 + col*p.rnum; 
      AzFloat *out = p.elm_out + base; 
      const AzFloat *bef = p.elm_bef + base;  
      const AzFloat *oneplussqavg = p.elm_oneplussqavg + base;       
      const AzFloat *grad = p.elm_grad + base;
    
      const int *whose_neigh = _column(col, p.whose_neighbor, p.nummax); 

      AzFloat val = 0; 
      if (bef[row] != 0) {
        int ix; 
        for (ix = 0; ix < p.nummax; ++ix) {    
          int kx = whose_neigh[ix]; 
          if (kx == p.stopper) break;         
          val += _entry(row, kx, tmp, p.rnum); 
        }       
      }
      out[row] = bef[row]*val + grad[row]*pow(oneplussqavg[row], -p.beta);  
    }
  }

  /*------------------------------------------------------------*/
  void azccall_unresnorm_local(const azcparam_unresnorm_local p) {
    int num = p.data_num * p.cnum * p.rnum; 
    int bb, tt; 
    azc_config(num, bb, tt, "azccall_unresnorm_local"); 
    azc_kernel(azc_unresnorm_local,bb,tt)(num, p); 
    chk_err("azccall_unresnorm_local", bb, tt);         
  } 

  /*------------------------------------------------*/
  /* loss: f(p)=log(1+exp(-zp)) for y in {0,1}      */
  /* derivative: f'(p)=-z/(exp(zp)+1)               */
  /* z=2y-1 for y in {0,1}; z=y for y in {-1,1}     */                          
  /*------------------------------------------------*/
  __global__ void azc_binlogi_deriv(bool is_01, 
             const AzFloat *p, const AzFloat *y, int num, 
             AzFloat *ld, AzFloat *loss) {
    for (int ex = azc_thno; ex < num; ex += azc_thnum) {    
      AzFloat pp = p[ex]; 
      AzFloat yy = (is_01) ? 2*y[ex]-1 : y[ex]; 
      AzFloat ee = exp(yy*pp); 
      ld[ex] = -yy/(ee+1); /* -z/(exp(zp)+1) */
      if (loss != NULL) {
        loss[ex] = log(1+1/ee);  /* log(1+exp(-zp)) */
      }
    }
  }
  /*------------------------------------------------------------*/
  void azccall_binlogi_deriv(bool is_01, 
             const AzFloat *p, const AzFloat *y, int num, 
             AzFloat *ld, AzFloat *loss) {
    int bb, tt; 
    azc_config(num, bb, tt, "azccall_binlogi_deriv"); 
    azc_kernel(azc_binlogi_deriv,bb,tt)(is_01, p, y, num, ld, loss); 
    chk_err("azccall_binlogi_deriv", bb, tt);         
  } 
  /*------------------------------------------------------------*/  
  __global__ void azc_binlogi_loss(bool is_01, 
             const AzFloat *p, const AzFloat *y, int num, 
             AzFloat *loss) {
    for (int ex = azc_thno; ex < num; ex += azc_thnum) {    
      AzFloat pp = p[ex]; 
      AzFloat yy = (is_01) ? 2*y[ex]-1 : y[ex]; 
      loss[ex] = log(1+exp(-yy*pp));  /* log(1+exp(-zp)) */
    }
  }  
  /*------------------------------------------------------------*/
  void azccall_binlogi_loss(bool is_01, 
             const AzFloat *p, const AzFloat *y, int num, 
             AzFloat *loss) {
    int bb, tt; 
    azc_config(num, bb, tt, "azccall_binlogi_loss"); 
    azc_kernel(azc_binlogi_loss,bb,tt)(is_01, p, y, num, loss); 
    chk_err("azccall_binlogi_loss", bb, tt);         
  } 
  
  /*------------------------------------------------------------*/  
  __global__ void azc_for_log_loss(const AzFloat *p, const int *y_row,
                                   int rnum, int cnum, AzFloat *out) {
    for (int col = azc_thno; col < cnum; col += azc_thnum) out[col] = _entry(y_row[col], col, p, rnum); 
  }  
  /*------------------------------------------------------------*/
  void azccall_for_log_loss(const AzFloat *p, const int *y_row, 
                            int rnum, int cnum, AzFloat *out) {                     
    int bb, tt; azc_config(cnum, bb, tt, "azccall_for_log_loss"); 
    azc_kernel(azc_for_log_loss,bb,tt)(p, y_row, rnum, cnum, out); 
    chk_err("azccall_for_log_loss", bb, tt);         
  } 
  /*------------------------------------------------------------*/  
  __global__ void azc_for_log_grad(AzFloat *p, const int *y_row, 
                                   int rnum, int cnum) {
    for (int col = azc_thno; col < cnum; col += azc_thnum) _entry(y_row[col], col, p, rnum) -= 1; 
  }  
  /*------------------------------------------------------------*/
  void azccall_for_log_grad(AzFloat *p, const int *y_row, 
                            int rnum, int cnum) {                     
    int bb, tt; azc_config(cnum, bb, tt, "azccall_for_log_grad"); 
    azc_kernel(azc_for_log_grad,bb,tt)(p, y_row, rnum, cnum); 
    chk_err("azccall_for_log_grad", bb, tt);         
  }   

  /*------------------------------------------------------------*/  
  /* normalize over columns so that the sum becomes one.  */
  __global__ void azc_sumone(AzFloat *inout, int rnum, const int *beg_end, int dnum, bool do_scale) {
    int num = rnum*dnum;             
    for (int ex = azc_thno; ex < num; ex += azc_thnum) {    
      int row = ex%rnum, dx = ex/rnum; 
      AzFloat sum = 0; 
      int bb = beg_end[dx*2], ee = beg_end[dx*2+1]; 
      int pos = bb*rnum+row; 
      for (int ix = bb; ix < ee; ++ix, pos+=rnum) sum += inout[pos]; 
      if (sum == 0) return; 
      if (do_scale) sum /= (AzFloat)(ee-bb); 
      pos = bb*rnum+row; 
      for (int ix = bb; ix < ee; ++ix, pos+=rnum) inout[pos]/= sum; 
    }
  }  
  /*------------------------------------------------------------*/
  void azccall_sumone(AzFloat *inout, int rnum, const int *beg_end, int dnum, bool do_scale) {
    int bb, tt; 
    azc_config(rnum*dnum, bb, tt, "azccall_sumone"); 
    azc_kernel(azc_sumone,bb,tt)(inout, rnum, beg_end, dnum, do_scale); 
    chk_err("azccall_sumone", bb, tt);         
  }

  /*------------------------------------------------------------*/  
  __global__ void azc_unsumone(AzFloat *grad, const AzFloat *inp, int rnum, const int *beg_end, int dnum, bool do_scale) {
    int num = rnum*dnum;             
    for (int ex = azc_thno; ex < num; ex += azc_thnum) {    
      int row = ex%rnum, dx = ex/rnum; 
      AzFloat sum = 0, zg = 0; 
      int bb = beg_end[dx*2], ee = beg_end[dx*2+1]; 
      int pos = bb*rnum+row; 
      for (int ix = bb; ix < ee; ++ix, pos+=rnum) {
        sum += inp[pos]; 
        zg += inp[pos]*grad[pos]; 
      }
      pos = bb*rnum+row;      
      AzFloat scale = (do_scale) ? ee-bb : 1; 
      if (sum != 0) { 
        AzFloat val0 = zg/sum/sum; 
        for (int ix = bb; ix < ee; ++ix, pos+=rnum) grad[pos] = scale*(grad[pos]/sum - val0); 
      }
    }
  }  
  /*------------------------------------------------------------*/
  void azccall_unsumone(AzFloat *grad, const AzFloat *inp, int rnum, const int *beg_end, int dnum, bool do_scale) {
    int bb, tt; 
    azc_config(rnum*dnum, bb, tt, "azccall_unsumone"); 
    azc_kernel(azc_unsumone,bb,tt)(grad, inp, rnum, beg_end, dnum, do_scale); 
    chk_err("azccall_unsumone", bb, tt);         
  }  
