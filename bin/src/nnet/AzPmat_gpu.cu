#include "hip/hip_runtime.h"
/* * * * *
 *  AzPmat_gpu.cu
 *  Copyright (C) 2013-2015,2017 Rie Johnson
 *
 *  This program is free software: you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation, either version 3 of the License, or
 *  (at your option) any later version.
 *
 *  This program is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License
 *  along with this program.  If not, see <http://www.gnu.org/licenses/>.
 * * * * */
 
#include "AzDmat.hpp" 
#include "AzPmat_gpu.cuh"
#include "AzMemTempl.hpp"
#include "AzCuda.cuh"

extern AzPdevice dev; 
extern int max_threads, max_blocks; 
#define cublas_handle dev.h

#ifdef __AZ_DOUBLE__  
/*---  double-precision  ---*/
#define cublasXgemm hipblasDgemm
#define cublasXaxpy hipblasDaxpy
#define cublasXamax hipblasIdamax
#define cublasXamin hipblasIdamin
#define curandGenUniform hiprandGenerateUniformDouble
#define curandGenNormal  hiprandGenerateNormalDouble
#define cublasXgeam hipblasDgeam
#define cublasXasum hipblasDasum
#define cublasXnrm2 hipblasDnrm2
#define cublasXdot hipblasDdot
#define cublasXscal hipblasDscal
#define cusolverDnXsyevd_bufferSize hipsolverDnDsyevd_bufferSize
#define cusolverDnXsyevd            hipsolverDnDsyevd
#define cusolverDnXsygvd_bufferSize hipsolverDnDsygvd_bufferSize
#define cusolverDnXsygvd            hipsolverDnDsygvd
#define cusolverDnXsyevj_bufferSize hipsolverDnDsyevj_bufferSize
#define cusolverDnXsyevj            hipsolverDnDsyevj
#else
/*---  single-precision  ---*/
#define cublasXgemm hipblasSgemm
#define cublasXaxpy hipblasSaxpy
#define cublasXamax hipblasIsamax
#define cublasXamin hipblasIsamin
#define curandGenUniform hiprandGenerateUniform
#define curandGenNormal  hiprandGenerateNormal 
#define cublasXgeam hipblasSgeam
#define cublasXasum hipblasSasum
#define cublasXnrm2 hipblasSnrm2
#define cublasXdot hipblasSdot
#define cublasXscal hipblasSscal
#define cusolverDnXsyevd_bufferSize hipsolverDnSsyevd_bufferSize
#define cusolverDnXsyevd            hipsolverDnSsyevd
#define cusolverDnXsygvd_bufferSize hipsolverDnSsygvd_bufferSize
#define cusolverDnXsygvd            hipsolverDnSsygvd
#define cusolverDnXsyevj_bufferSize hipsolverDnSsyevj_bufferSize
#define cusolverDnXsyevj            hipsolverDnSsyevj
#endif

/* pointers are all device pointers unless specified otherwise */

/*-------------------------------------------------------------*/
template <class T>
void _AzParr<T>::free() {
  if (elm != NULL) {  
    dev.pmem.free(no, elm, sizeof(T)*num);      
    elm = NULL; 
  }
  num = 0; 
}  
template void _AzParr<int>::free(); 
template void _AzParr<AzFloat>::free(); 
template void _AzParr<AzByte>::free(); 

/*-------------------------------------------------------------*/  
template <class T>
void _AzParr<T>::free_alloc(int inp_num, const char *str1, const char *str2) {
  free();  
  if (inp_num > 0) {
    size_t sz = sizeof(T)*inp_num; 
    elm = (T *)dev.pmem.alloc(no, sz, str1, str2);   
    num = inp_num; 
  }
  else if (inp_num < 0) {
    AzBytArr s(str1); s << " " << str2; 
    AzX::throw_if(true, "_AzParr::free_alloc", "negative area size -- possibly overflowing", s.c_str()); 
  }
}
template void _AzParr<int>::free_alloc(int, const char *, const char *); 
template void _AzParr<AzFloat>::free_alloc(int, const char *, const char *); 
template void _AzParr<AzByte>::free_alloc(int, const char *, const char *);

/*-------------------------------------------------------------*/  
void _AzPmat::_copy(AzFloat *dst, const AzFloat *src, int num, AzFloat coeff) 
{
  if (coeff != 1) azccall_copy(dst, src, num, coeff); 
  else            AzCuda::memcpy(dst, src, num*sizeof(src[0]), hipMemcpyDeviceToDevice, "_AzPmat::_copy"); 
}
  
/*-------------------------------------------------------------*/
void _AzPmat::_add_axpy(AzFloat *dst, const AzFloat *src, int num, AzFloat coeff) 
{
  if (coeff == 0 || num <= 0) return; 
  hipblasStatus_t ret = cublasXaxpy(cublas_handle, num, &coeff, src, 1, dst, 1); 
  AzCuda::throwIfblasError(ret, "_AzPmat::_add_axpy", "cublasXaxpy (addition) failed"); 
}

/*-------------------------------------------------------------*/
void _AzPmat::_multiply_scal(AzFloat *dst, AzFloat coeff, int num) 
{
  if (coeff == 1 || num <= 0) return; 
  hipblasStatus_t ret = cublasXscal(cublas_handle, num, &coeff, dst, 1); 
  AzCuda::throwIfblasError(ret, "_AzPmat::_multiply_scal", "cublasXscal (scaling) failed"); 
}
  
/*---  matrix product  ---*/             
/*-------------------------------------------------------------*/
void _AzPmat::_prod10(AzFloat *elm, int r_num, int c_num, 
                      const AzFloat *elm1, int row_num1,  
                      const AzFloat *elm2, int row_num2, 
                      int num,
                      const AzPstreams *streams,                      
                      AzFloat alpha, AzFloat beta) const
{
  if (r_num <= 0 || c_num <= 0) return; 

  if (streams != NULL) {
    streams->setStream(cublas_handle); 
  }
  /* C = alpha op(A) op(B) + beta C */
  hipblasStatus_t ret = cublasXgemm(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 
                    r_num, c_num, num, 
                    &alpha, 
                    elm1, row_num1, elm2, row_num2, &beta, 
                    elm, r_num); 
                    
  AzCuda::throwIfblasError(ret, "_AzPmat::_prod10", "hipblasSgemm (matrix multiplication) failed"); 
}                     

/*-------------------------------------------------------------*/
void _AzPmat::_prod01(AzFloat *elm, int r_num, int c_num, 
                     const AzFloat *elm1, int row_num1, 
                     const AzFloat *elm2, int row_num2,
                     int num,
                     const AzPstreams *streams, 
                     AzFloat alpha, AzFloat beta) const
{
  if (r_num <= 0 || c_num <= 0) return; 

  if (streams != NULL) {
    streams->setStream(cublas_handle); 
  }  
  hipblasStatus_t ret = cublasXgemm
                    (cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T, 
                    r_num, c_num, num, 
                    &alpha, 
                    elm1, row_num1, elm2, row_num2, &beta, 
                    elm, r_num);                 
  AzCuda::throwIfblasError(ret, "_AzPmat::_prod01", "hipblasSgemm (matrix multiplication) failed");          
}

/*-------------------------------------------------------------*/
void _AzPmat::_prod00(AzFloat *elm, int r_num, int c_num, 
                     const AzFloat *elm1, int row_num1, 
                     const AzFloat *elm2, int row_num2, 
                     int num,
                     const AzPstreams *streams, 
                     AzFloat alpha, AzFloat beta) const
{
  if (r_num <= 0 || c_num <= 0) return; 

  if (streams != NULL) {
    streams->setStream(cublas_handle); 
  }  
  hipblasStatus_t ret = cublasXgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                    r_num, c_num, num, 
                    &alpha, 
                    elm1, row_num1, elm2, row_num2, &beta, 
                    elm, r_num); 
                   
  AzCuda::throwIfblasError(ret, "_AzPmat::_prod00", "hipblasSgemm (matrix multiplication) failed");     
}                     

/*-------------------------------------------------------------*/
double _AzPmat::_absmax(const AzFloat *elm, int num, int *out_index) const 
{
  const char *eyec = "_AzPmat::_absmax"; 
  int index; 
  hipblasStatus_t ret = cublasXamax(cublas_handle, num, elm, 1, &index);         
  AzCuda::throwIfblasError(ret, eyec, "cublasXamax failed");     
  
  AzFloat h_val; 
  hipError_t ret2 = hipMemcpy(&h_val, elm+index-1, sizeof(AzFloat), hipMemcpyDeviceToHost); 
  AzCuda::throwIfError(ret2, eyec); 
  if (out_index != NULL) *out_index = index; 
  return (double)h_val; 
} 

/*-------------------------------------------------------------*/
double _AzPmat::_absmin(const AzFloat *elm, int num, int *out_index) const 
{
  const char *eyec = "_AzPmat::_absmin"; 
  int index; 
  hipblasStatus_t ret = cublasXamin(cublas_handle, num, elm, 1, &index);         
  AzCuda::throwIfblasError(ret, eyec, "cublasXamin failed");     
  
  AzFloat h_val; 
  hipError_t ret2 = hipMemcpy(&h_val, elm+index-1, sizeof(AzFloat), hipMemcpyDeviceToHost); 
  AzCuda::throwIfError(ret2, eyec); 
  if (out_index != NULL) *out_index = index; 
  return (double)h_val; 
} 

/*-------------------------------------------------------------*/
/* faster: 3.18 vs. 121.5 */
AzFloat _AzPmat::_absSum_cublas(const AzFloat *elm, int num)
{
  const char *eyec = "_AzPmat::_abssum_cublas"; 
  AzFloat asum = 0; 
  hipblasStatus_t ret = cublasXasum(cublas_handle, num, elm, 1, &asum);         
  AzCuda::throwIfblasError(ret, eyec, "cublasXasum failed"); 
  return asum; 
}

/*-------------------------------------------------------------*/
/* faster : 3.43 vs. 121.58 */
AzFloat _AzPmat::_norm2_cublas(const AzFloat *elm, int num)
{
  const char *eyec = "_AzPmat::_norm2_cublas"; 
  AzFloat nrm2 = 0; 
  hipblasStatus_t ret = cublasXnrm2(cublas_handle, num, elm, 1, &nrm2);         
  AzCuda::throwIfblasError(ret, eyec, "cublasXnrm2 failed"); 
  return nrm2; 
}

/*-------------------------------------------------------------*/
AzFloat _AzPmat::_sum_cublas(const AzFloat *elm, int num)
{
  _AzParr<AzFloat> val; val.free_alloc(1); 
  AzFloat one = 1; 
  val.copy_from_host(&one, 1); 
  AzFloat sum = 0; 
  hipblasStatus_t ret = cublasXdot(cublas_handle, num, elm, 1, val._dptr(), 0, &sum); 
  AzCuda::throwIfblasError(ret, "_AzPmat::_sum_dot", "cublasXdot failed"); 
  return sum; 
} 
 
/*---------------------------------------------------------------*/  
void _AzPmat::sh_config(int num, int &bb, int &tt, const char *msg) {
  AzX::throw_if((num <= 0), msg, "_AzPmat::sh_cofig, num must be positive"); 
  tt = MIN(num, MIN(max_threads, azc_numShared)); 
  bb = MIN((num+tt-1)/tt, max_blocks);    
} 
 
/*-------------------------------------------------------------*/
AzFloat _AzPmat::_get_sum(int op, const AzFloat *src, int num) 
{
  if (num <= 0) return 0; 
  int bb, tt; 
  sh_config(num, bb, tt, "_get_sum");
  _AzParr<AzFloat> o; 
  o.free_alloc(bb, "_AzPmat::_get_sum");  /* device memory */
  azc_kernel(azcsh_sum,bb,tt)(op, src, num, o._dptr_u()); 
  AzCuda::check_error("_AzPmat::_get_sum", bb, tt); 
  
  AzFloat *arr = NULL;     
  AzBaseArray<AzFloat> a_arr; 
  a_arr.alloc(&arr, bb, "_AzPmat::_get_sum", "arr");  /* host memory */
  o.copy_to_host(arr, bb); 
  AzFloat sum = 0; 
  for (int ix = 0; ix < bb; ++ix) {
    sum += arr[ix]; 
  }
  return sum; 
}

/*-------------------------------------------------------------*/
int _AzPmat::_nz(const AzFloat *src, int num) 
{
  if (num <= 0) return 0; 
  int bb, tt; 
  sh_config(num, bb, tt, "_nz");
  _AzParr<int> o; 
  o.free_alloc(bb, "_AzPmat::_nz");  /* device memory */
  azc_kernel(azcsh_nz,bb,tt)(src, num, o._dptr_u()); 
  AzCuda::check_error("_AzPmat::_nz", bb, tt); 
  
  AzIntArr ia; 
  ia.reset(bb, 0); 
  o.copy_to_host(ia.point_u(), bb); 
  int sum = ia.sum(); 
  return sum; 
}

/*-------------------------------------------------------------*/
double _AzPmat::_min(const AzFloat *src, int num, int *out_index) 
{
  if (num <= 0) {
    if (out_index != NULL) *out_index = -1; 
    return 0; 
  }
  int bb, tt; 
  sh_config(num, bb, tt, "_min");
  _AzParr<int> _ind; 
  _ind.free_alloc(bb, "_AzPmat::_min,_ind");  /* device memory */
  _AzParr<double> _val; 
  _val.free_alloc(bb, "_AzPmat::_min,_val");  /* device memory */
  azc_kernel(azcsh_min,bb,tt)(src, num, _ind._dptr_u(), _val._dptr_u()); 
  AzCuda::check_error("_AzPmat::_min", bb, tt); 

  AzDvect v_val(bb); 
  _val.copy_to_host(v_val.point_u(), bb); 
  
  int index; 
  double val = v_val.min(&index); 
  if (out_index != NULL) {
    hipError_t ret = hipMemcpy(out_index, _ind._dptr() + index, sizeof(int), hipMemcpyDeviceToHost); 
    AzCuda::throwIfError(ret, "_AzPmat::_min,copying index");   
  }
  return val; 
}  

/*-------------------------------------------------------------*/
double _AzPmat::_max(const AzFloat *src, int num, int *out_index) 
{
  if (num <= 0) {
    if (out_index != NULL) *out_index = -1; 
    return 0; 
  }
  int bb, tt; 
  sh_config(num, bb, tt, "_max");
  _AzParr<int> _ind; 
  _ind.free_alloc(bb, "_AzPmat::_max,_ind");  /* device memory */
  _AzParr<double> _val; 
  _val.free_alloc(bb, "_AzPmat::_max,_val");  /* device memory */
  azc_kernel(azcsh_max,bb,tt)(src, num, _ind._dptr_u(), _val._dptr_u()); 
  AzCuda::check_error("_AzPmat::_max", bb, tt); 

  AzDvect v_val(bb); 
  _val.copy_to_host(v_val.point_u(), bb); 
  
  int index; 
  double val = v_val.max(&index); 
  if (out_index != NULL) {
    hipError_t ret = hipMemcpy(out_index, _ind._dptr() + index, sizeof(int), hipMemcpyDeviceToHost); 
    AzCuda::throwIfError(ret, "_AzPmat::_max,copying index");   
  }
  return val; 
}  

/*-------------------------------------------------------------*/
void _AzPmat::_add_colSum(int op, const AzFloat *src, int row_num, int col_num, 
                             AzFloat *col_sum) 
{
  if (row_num <= 0 || col_num <= 0) return; 
  int tt = MIN(row_num, MIN(max_threads, azc_numShared));   
  for (int cx = 0; cx < col_num; cx += max_blocks) {
    int bb = MIN(max_blocks, col_num-cx); 
    azc_kernel(azcsh_add_colSum,bb,tt)(op, src+cx*row_num, row_num, col_num-cx, col_sum+cx); 
    AzCuda::check_error("_AzPmat::_get_colSum", bb, tt); 
  }
}

/*-------------------------------------------------------------*/
void _AzPrng::uniform_01(AzFloat *dev_data, size_t sz) 
{
  AzX::throw_if((!is_rg_set), "_AzPrng::uniform_01", "not ready"); 
  AzX::throw_if((dev_data == NULL), "_AzPrng::uniform", "null pointer"); 
  if (sz <= 0) return; 
  hiprandStatus_t ret = curandGenUniform(rg, dev_data, sz); 
}

/*-------------------------------------------------------------*/
void _AzPrng::normal(AzFloat *dev_data, int sz, AzFloat mean, AzFloat sdev) {
  AzX::throw_if((!is_rg_set), "_AzPrng::normal", "not ready"); 
  AzX::throw_if((dev_data == NULL), "_AzPrng::normal", "null pointer"); 
  if (sz <= 0) return; 
  hiprandStatus_t ret = curandGenNormal(rg, dev_data, (size_t)sz, mean, sdev); 
}

/*-------------------------------------------------------------*/
void _AzPmat::_transpose_cublas(const AzFloat *src, int r_num, int c_num, AzFloat *dst) 
{
  AzFloat alpha = 1, beta = 0; 
  /* C = alpha op(A) + beta op(B) */
  hipblasStatus_t ret = 
  cublasXgeam(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 
              c_num, r_num,  /* dimension of the result */
              &alpha, src, r_num, 
              &beta, src, c_num,  /* dummy ptr */
              dst, c_num); 
  AzCuda::throwIfblasError(ret, "_AzPmat::_transpose_cublas", "cublasXgeam (for transpose) failed");               
}

#ifdef __AZ_EIGEN__
/*-------------------------------------------------------------*/
/*-------------------------------------------------------------*/
void _AzPmat::_dbl_sym_eigen(double *inout, /* input: A (symmetric), output: eigenvectors */
                     int rc_num, 
                     double *lambdas, /* output: eigenvalues */
                     bool do_valueonly,                      
                     int *devinfo) /* device */ {                    
  if (rc_num <= 0) return; 
  const char *eyec = "_AzPmat::_dbl_sym_eigen";  
  hipsolverHandle_t solver; 
  hipsolverStatus_t ret = hipsolverDnCreate(&solver); 
  AzCuda::throwIfsolverError(ret, eyec, "hipsolverDnCreate"); 
  
  hipsolverEigMode_t jobz = (do_valueonly) ? HIPSOLVER_EIG_MODE_NOVECTOR : HIPSOLVER_EIG_MODE_VECTOR; 
  hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_UPPER; 
  int lwork = 0; 
  ret = hipsolverDnDsyevd_bufferSize(solver, jobz, uplo, 
                                    rc_num, inout, rc_num, lambdas, &lwork); 
  AzCuda::throwIfsolverError(ret, eyec, "hipsolverDnDsyevd_bufferSize failed"); 
  _AzParr<double> work; work.free_alloc(lwork, eyec, "lwork"); /* work area */
    
  ret = hipsolverDnDsyevd(solver, jobz, uplo, rc_num, inout, rc_num, lambdas, 
                         work._dptr_u(), lwork, devinfo);
  AzCuda::throwIfsolverError(ret, eyec, "hipsolverDnDsyevd (eigen computation) failed");     

  ret = hipsolverDnDestroy(solver); 
  AzCuda::throwIfsolverError(ret, eyec, "hipsolverDnDestroy");   
}

/*-------------------------------------------------------------*/
/*
 *  Ax = lam x if B=NULL
 *  Ax = lam Bx if B!=NULL && do_AxBx==true
 *  ABx = lam x if B!=NULL && do_AxBx==false
 */
void _AzPmat::_sym2_eigen(AzFloat *inoutA, /* input: A (symmetric), output: eigenvectors */
                     AzFloat *inoutB,
                     int rc_num,        /* #rows=#cols */
                     bool do_AxBx,      /* true: Ax=lam Bx, false: ABx=lam x */
                     AzFloat *lambdas,  /* output: eigenvalues */
                     bool do_valueonly,                      
                     int *devinfo)      /* device */ {                    
  if (rc_num <= 0) return; 
  const char *eyec = "_AzPmat::sym2_eigen";  
  hipsolverHandle_t solver; 
  hipsolverStatus_t ret = hipsolverDnCreate(&solver); 
  AzCuda::throwIfsolverError(ret, eyec, "hipsolverDnCreate"); 
  hipsolverEigMode_t jobz = (do_valueonly) ? HIPSOLVER_EIG_MODE_NOVECTOR : HIPSOLVER_EIG_MODE_VECTOR; 
  hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_UPPER;
  
  if (inoutB == NULL) {
    int lwork = 0; 
    ret = cusolverDnXsyevd_bufferSize(solver, jobz, uplo, 
                                      rc_num, inoutA, rc_num, lambdas, &lwork); 
    AzCuda::throwIfsolverError(ret, eyec, "hipsolverDnSsyevd_bufferSize failed"); 
    _AzParr<AzFloat> work; work.free_alloc(lwork, eyec, "lwork"); /* work area */
    
    ret = cusolverDnXsyevd(solver, jobz, uplo, rc_num, inoutA, rc_num, lambdas, 
                           work._dptr_u(), lwork, devinfo);
    AzCuda::throwIfsolverError(ret, eyec, "hipsolverDnSsyevd (eigen computation) failed");    
  }
  else {
    hipsolverEigType_t  eigtyp = (do_AxBx) ? HIPSOLVER_EIG_TYPE_1 : HIPSOLVER_EIG_TYPE_2; 
    int lwork = 0; 
    ret = cusolverDnXsygvd_bufferSize(solver, eigtyp, jobz, uplo, 
                                      rc_num, inoutA, rc_num, inoutB, rc_num, lambdas, &lwork); 
    AzCuda::throwIfsolverError(ret, eyec, "hipsolverDnSsygvd_bufferSize failed"); 
    _AzParr<AzFloat> work; work.free_alloc(lwork, eyec, "lwork"); /* work area */
    
    ret = cusolverDnXsygvd(solver, eigtyp, jobz, uplo, rc_num, inoutA, rc_num, inoutB, rc_num, lambdas, 
                           work._dptr_u(), lwork, devinfo);
    AzCuda::throwIfsolverError(ret, eyec, "hipsolverDnSsygvd (eigen computation) failed");     
  }

  ret = hipsolverDnDestroy(solver); 
  AzCuda::throwIfsolverError(ret, eyec, "hipsolverDnDestroy");   
}

#ifdef __AZ_CUDA9__
/*-------------------------------------------------------------*/
void _AzPmat::_symj_eigen(AzFloat *inoutA, /* input: A (symmetric), output: eigenvectors */
                     int rc_num,        /* #rows=#cols */
                     AzFloat *lambdas,  /* output: eigenvalues */
                     bool do_valueonly,                      
                     int *devinfo)      /* device */ {                    
  if (rc_num <= 0) return; 
  const char *eyec = "_AzPmat::symj_eigen";  
  hipsolverHandle_t solver; 
  hipsolverStatus_t ret = hipsolverDnCreate(&solver); 
  AzCuda::throwIfsolverError(ret, eyec, "hipsolverDnCreate"); 
  hipsolverEigMode_t jobz = (do_valueonly) ? HIPSOLVER_EIG_MODE_NOVECTOR : HIPSOLVER_EIG_MODE_VECTOR; 
  hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_UPPER;

  hipsolverSyevjInfo_t params; 
  ret = hipsolverDnCreateSyevjInfo(&params);
  AzCuda::throwIfsolverError(ret, eyec, "hipsolverDnCreateSyevjInfo"); 
  
  int lwork = 0; 
  ret = cusolverDnXsyevj_bufferSize(solver, jobz, uplo, 
                                    rc_num, inoutA, rc_num, lambdas, &lwork, params); 
  AzCuda::throwIfsolverError(ret, eyec, "hipsolverDnSsyevj_bufferSize failed"); 
  _AzParr<AzFloat> work; work.free_alloc(lwork, eyec, "lwork"); /* work area */
    
  ret = cusolverDnXsyevj(solver, jobz, uplo, rc_num, inoutA, rc_num, lambdas, 
                         work._dptr_u(), lwork, devinfo, params);
  AzCuda::throwIfsolverError(ret, eyec, "hipsolverDnSsyevj (eigen computation) failed");    

  ret = hipsolverDnDestroySyevjInfo(params);
  AzCuda::throwIfsolverError(ret, eyec, "hipsolverDnDestroySyevjInfo");   
  
  ret = hipsolverDnDestroy(solver); 
  AzCuda::throwIfsolverError(ret, eyec, "hipsolverDnDestroy");   
}
#endif 
#endif 
